#include <stdio.h>
#include <hip/hip_runtime.h>

#define ARRAY_SIZE 8
#define BLOCK_SIZE 16

__global__ void preSum(int *g_num, int *ans,int size)
{
	int tx = threadIdx.x;
	int pOut = 0,pIn = 1;
	__shared__ int s_num[ARRAY_SIZE*2];
	
	s_num[tx] = s_num[tx + size] = g_num[tx];

	__syncthreads();

	for(int offset = 1; offset < size; offset *= 2)
	{
		pOut = 1 - pOut;
		pIn = 1 - pOut;
		if(tx >= offset)
			s_num[pOut * size + tx] = s_num[pIn * size + tx] + s_num[pIn * size + tx - offset];
		else
			s_num[pOut * size + tx] = s_num[pIn * size + tx];
		__syncthreads();
	}
	ans[tx] = s_num[pOut * size + tx];
}

int main()
{
	int num[8] = {3,1,7,0,4,1,6,3};
	int *h_num = (int*)malloc(ARRAY_SIZE);
	int *h_ans = (int*)malloc(ARRAY_SIZE);
	int *d_num, *d_ans;
	int *arr;
	hipMalloc((void **)&d_num,sizeof(int) * ARRAY_SIZE);
	hipMalloc((void **)&d_ans,sizeof(int) * ARRAY_SIZE);
	
	for(int i = 0; i < ARRAY_SIZE; ++i)
		h_num[i] = num[i];
	for(int i = 0; i < ARRAY_SIZE; ++i)
		printf("%d\n",h_num[i]);

	hipMemcpy(d_num,h_num,sizeof(int) * ARRAY_SIZE,hipMemcpyHostToDevice);
	
	preSum<<<1,BLOCK_SIZE>>>(d_num,d_ans,ARRAY_SIZE);
	
	hipMemcpy(h_ans,d_ans,sizeof(int) * ARRAY_SIZE,hipMemcpyDeviceToHost);

	for(int i = 0; i < ARRAY_SIZE; ++i)
		printf("i %d %d\n",i,h_ans[i]);
	printf("\n");

//	free(h_num);
//	free(h_ans);
//	cudaFree(d_num);
//	cudaFree(d_ans);
	return 0;
}

