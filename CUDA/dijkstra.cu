
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cstring>
using namespace std;

#define N 1024
#define INF 0x03F3F3F
int map[N * N];
int path[N];

__global__ void kernel1(int *nv,int *map,int *cost,int *tmp_cost,bool *visited,int * tmp_path)
{
	const int nn = 1024;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//fetch all its neighbour
	if(visited[tid])
	{
		visited[tid] = false;
		for(int i = 0; i < *nv; ++i)
		{
			if(cost[tid] + map[tid * nn + i] < tmp_cost[i])
			{
				tmp_path[i] = tid;
				tmp_cost[i] = cost[tid] + map[tid * N + i];
			}
		}
	}
	return;
}
__global__ void kernel2(int *cost,int *tmp_cost,bool *visited,bool *flag,int *tmp_path,int *path)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//some vertex can be updated,means that it is the new one to the queue
	if(cost[tid] > tmp_cost[tid])
	{
		path[tid] = tmp_path[tid];
		visited[tid] = true;
		cost[tid] = tmp_cost[tid];
		*flag = true;
	}
	tmp_cost[tid] = cost[tid];//if cost[tid] <= tmp_cost[tid]
	tmp_path[tid] = path[tid];
	return;
}
void print_path(int node,int path[])
{
	if(path[node] == -1)
	{
		printf("%d ",node + 1);
		return;
	}
	else
	{
		print_path(path[node],path);
		printf("%d ",node + 1);
	}
	return;
}

int main()
{
	freopen("input_dijkstra","r",stdin);
	int cost[N],tmp_cost[N],path[N],tmp_path[N];
	bool visited[N],flag;

	int *dev_cost,*dev_tmp_cost,*dev_map,*dev_nv,*dev_path,*dev_tmp_path;
	bool *dev_visited,*dev_flag;

	int nv,ne;
	scanf("%d%d",&nv,&ne);
	//initialize
	for(int i = 0; i < nv; ++i)
	{
		cost[i] = INF;
		tmp_cost[i] = INF;
		visited[i] = false;
		for(int j = 0; j < nv; ++j)
			map[i * N + j] = map[j * N + i] = INF;
	}
	//read all the edges
	for(int i = 0; i < ne; ++i)
	{
		int p,q,w;
		scanf("%d%d%d",&p,&q,&w);
		map[p * N + q] = map[q * N + p] = w;
	}

	int source;
	scanf("%d",&source);
	cost[source] = tmp_cost[source] = 0;
	visited[source] = true;
	flag = true;
	path[source] = -1;
	tmp_path[source] = -1;

	cout<<0<<endl;
	hipMalloc((void**)&dev_cost,N * sizeof(int));
	cout<<1<<endl;
	hipMalloc((void**)&dev_tmp_cost,N * sizeof(int));
	hipMalloc((void**)&dev_visited,N * sizeof(bool));
	hipMalloc((void**)&dev_map,N * N * sizeof(int));
	hipMalloc((void**)&dev_flag,sizeof(bool));
	hipMalloc((void**)&dev_nv,sizeof(int));
	hipMalloc((void**)&dev_path,N * sizeof(int));
	hipMalloc((void**)&dev_tmp_path,N * sizeof(int));

	hipMemcpy(dev_cost,cost,N * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_tmp_cost,tmp_cost,N * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_visited,visited,N * sizeof(bool),hipMemcpyHostToDevice);
	hipMemcpy(dev_map,map,N * N * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_flag,&flag,sizeof(bool),hipMemcpyHostToDevice);
	hipMemcpy(dev_nv,&nv,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_path,path,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_tmp_path,tmp_path,N*sizeof(int),hipMemcpyHostToDevice);

	while(flag)
	{
		flag = false;
		kernel1<<<1,nv>>>(dev_nv,dev_map,dev_cost,dev_tmp_cost,dev_visited,dev_tmp_path);
		hipMemcpy(dev_flag,&flag,sizeof(bool),hipMemcpyHostToDevice);
		kernel2<<<1,nv>>>(dev_cost,dev_tmp_cost,dev_visited,dev_flag,dev_tmp_path,dev_path);
		hipMemcpy(&flag,dev_flag,sizeof(bool),hipMemcpyDeviceToHost);
	}
	hipMemcpy(cost,dev_cost,N*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(path,dev_path,N*sizeof(int),hipMemcpyDeviceToHost);
	for(int i = 0; i < nv; ++i)
	{
		printf("Node %d cost = %2d  path: ",i+1,cost[i]);
		print_path(i,path);
		printf("\n");
	}
	cout << endl;
	return 0;
}



