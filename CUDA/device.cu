
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
using namespace std;

int main()
{
	hipDeviceProp_t prop;
	int count;
	hipGetDeviceCount(&count);
	cout << count << endl;
	for(int i = 0; i < count;++i)
	{
		hipGetDeviceProperties(&prop,i);
		printf("Name: %s\n",prop.name);
		printf("Cumpute capability: %d.%d\n",prop.major,prop.minor);
		printf("\n");
	}
}

