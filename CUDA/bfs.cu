
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cstring>
using namespace std;
#define V 1024
#define E 1024
struct NODE
{
	int begin;
	int end;
}node[V];
__global__ void bfs(NODE * node,int * edge,int * cost,bool * frontier,bool * visited)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(frontier[tid])
	{
		frontier[tid] = false;
		visited[tid] = true;
		for(int i = node[tid].begin; i < node[tid].end; ++i)
		{
			if(!visited[edge[i]])
			{
				cost[edge[i]] = cost[tid] + 1;
				frontier[edge[i]] = true;
			}
		}
	}
	return;
}
int main()
{
	freopen("input_bfs","r",stdin);
	int nv,ne;
	scanf("%d%d",&nv,&ne);

	int edge[2*E],cost[V];
	bool frontier[V],visited[V];

	int *dev_edge,*dev_cost;
	bool *dev_frontier,*dev_visited;
	NODE *dev_node;
	memset(cost,-1,nv * sizeof(int));
	memset(frontier,false,nv * sizeof(bool));
	memset(visited,false,nv * sizeof(bool));

	for(int i = 0; i < nv;++i)
	{
		int edge_num;
		scanf("%d",&edge_num);
		if(i == 0)
			node[i].begin = 0;
		else
			node[i].begin = node[i-1].end;
		node[i].end = node[i].begin;
		//read edges of this vertex
		for(int j = 0; j < edge_num; ++j)
		{
			int vertex;
			scanf("%d",&vertex);
			edge[node[i].begin + j] = vertex;
			node[i].end++;
		}
	}
	for(int i = 0; i < 2*ne; ++i)
		cout << edge[i] << " ";
	cout << endl;

	hipMalloc((void**)&dev_edge,2*ne*sizeof(int));
	hipMalloc((void**)&dev_cost,nv*sizeof(int));
	hipMalloc((void**)&dev_frontier,nv*sizeof(bool));
	hipMalloc((void**)&dev_visited,nv*sizeof(bool));
	hipMalloc((void**)&dev_node,nv*sizeof(NODE));

	hipMemcpy(dev_edge,edge,2*ne*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_visited,visited,nv*sizeof(bool),hipMemcpyHostToDevice);
	hipMemcpy(dev_node,node,nv*sizeof(NODE),hipMemcpyHostToDevice);

	int source;
	scanf("%d",&source);
	frontier[source] = true;
	cost[source] = 0;
	hipMemcpy(dev_cost,cost,nv*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_frontier,frontier,nv*sizeof(bool),hipMemcpyHostToDevice);
	bool flag = true;
	while(flag)
	{
		flag = false;
		for(int i = 0; i < nv;++i)
		{
			if(frontier[i])
			{
				cout << "frontier " << i << endl;
				flag = true;
				bfs<<<1,nv>>>(dev_node,dev_edge,dev_cost,dev_frontier,dev_visited);
				hipMemcpy(frontier,dev_frontier,nv*sizeof(bool),hipMemcpyDeviceToHost);
				break;
			}
		}
	}
	hipMemcpy(cost,dev_cost,nv*sizeof(int),hipMemcpyDeviceToHost);
	for(int i = 0; i < nv; ++i)
		printf("%d %d\n",i,cost[i]);
	return 0;
}






