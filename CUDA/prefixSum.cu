#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define ARRAY_SIZE 8
#define BLOCK_SIZE 16

__global__ void preSum(int *g_num, int *ans,int size)
{
	int tx = threadIdx.x;
	int pOut = 0,pIn = 1;
	__shared__ int s_num[2][BLOCK_SIZE];
	
	s_num[0][tx] = g_num[tx];
	s_num[1][tx] = s_num[0][tx];

	__syncthreads();

	for(int offset = 1; offset < size; offset *= 2)
	{
		pOut = 1 - pOut;
		pIn = 1 - pOut;
		if(tx >= offset)
			s_num[pOut][tx] = s_num[pIn][tx] + s_num[pIn][tx - offset];
		else
			s_num[pOut][tx] = s_num[pIn][tx];
		__syncthreads();
	}
	ans[tx] = s_num[pOut][tx];
}

int main()
{
	int num[8] = {3,1,7,0,4,1,6,3};
	int *h_num = (int*)malloc(ARRAY_SIZE);
	int *h_ans = (int*)malloc(ARRAY_SIZE);
	int *d_num, *d_ans;
	hipMalloc((void **)&d_num,sizeof(int) * ARRAY_SIZE);
	hipMalloc((void **)&d_ans,sizeof(int) * ARRAY_SIZE);
	
	for(int i = 0; i < ARRAY_SIZE; ++i)
		h_num[i] = num[i];
		printf("%d\n",h_num[i]);

	hipMemcpy(d_num,h_num,sizeof(int) * ARRAY_SIZE,hipMemcpyHostToDevice);
	
	preSum<<<1,BLOCK_SIZE>>>(d_num,d_ans,ARRAY_SIZE);
	
	hipMemcpy(h_ans,d_ans,sizeof(int) * ARRAY_SIZE,hipMemcpyDeviceToHost);

	for(int i = 0; i < ARRAY_SIZE; ++i)
		printf("i %d %d\n",i,h_ans[i]);
	printf("\n");

//	free(h_num);
//	free(h_ans);
//	hipFree(d_num);
//	hipFree(d_ans);
	return 0;
}

